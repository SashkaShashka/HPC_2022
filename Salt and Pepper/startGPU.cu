#include "hip/hip_runtime.h"
#include "EasyBMP.h"
#include <iostream>
#include <vector>
#include <algorithm> 
#include <string>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void kernel(float* arrayOutput, hipTextureObject_t texObj, int width, int height) {

	int index_x = blockIdx.x * blockDim.x + threadIdx.x;
	int index_y = blockIdx.y * blockDim.y + threadIdx.y;
	int array[9];
	int k = 0;
	if ((index_x < width) && (index_y < height))
	{
		for (int i = index_x-1; i <= index_x + 1; i++)
		{
			for (int j = index_y-1; j <= index_y + 1; j++)
			{
				array[k] = (int)tex2D<float>(texObj, i, j);
				k++;
			}
		}
		for (int q = 0; q < 9; q++) 
		{
        		for (int w = 0; w < 8; w++) 
			{
            			if (array[w] > array[w + 1]) 
				{
               				float b = array[w]; // ������� �������������� ����������
                			array[w] = array[w+ 1]; // ������ �������
               				array[w + 1] = b; // �������� ���������
            			}
        		}
    		}
		arrayOutput[(index_x)  + (index_y)* (width)] = array[4];

	}
}

void moveCursor(std::ostream& os, int col, int row)
{
  os << "\033[" << col << ";" << row << "H";
}

void draw_frame() {
	// ����� ������ ������� ����� ������� �����, ������� 201, �.�. ����������� ����������� ����������� ASCII � ���� ������� ���
	cout << char(201);
	for (size_t i = 0; i < 100; i++)
		cout << char(201);	
	cout << char(201);
	cout << endl;
	cout << char(201);
	for (size_t i = 0; i < 49; i++)
		cout << "-";
	cout << "0%";
	for (size_t i = 0; i < 49; i++)
		cout << "-";
	cout << char(201);
	moveCursor(std::cout,3,1);
	cout << char(201);
	for (size_t i = 0; i < 100; i++)
	 	cout << char(201);
	cout << char(201);
	cout << endl;
}

void PrintProcent (int procent) {
	moveCursor(std::cout, 2, 2);
	for (size_t i = 0; i < procent; i++)
		cout << "|";	
	for (size_t i = 0; i < 100-(procent); i++)
		cout << "-";	
	cout << char(186);
	moveCursor(std::cout, 2,49);
	string procent_str = to_string(procent);
	if (procent < 49) cout << procent_str << "%";
	if (procent == 49) cout << procent_str[0] << procent_str[1] << "%";
	if (procent == 50) cout << procent_str << "%";
	if (procent > 50) cout << procent_str << "%";	cout << endl << endl;

}

int MedianFilter(vector<vector<int>> image, int verctical, int gorizontal) {
	vector<int> array;

	for (int i = verctical - 1; i <= verctical + 1; i++)
	{
		for (int j = gorizontal - 1; j <= gorizontal + 1; j++)
		{
			array.push_back(image[i][j]);
		}
	}
	sort(array.begin(), array.end());
	return array[4];
}

vector<vector<int>> transformationImage(vector<vector<int>> image) {
	vector<vector<int>> output(image.size(), vector <int>(image[0].size()));
	int last_procent = -1;

	//float count = 0;
	//float h = image.size()/100;
	for (int i = 1; i < image.size() - 1; i++)
	{
		int procent = (int)(((float)i / (float)(image.size() - 2)) * 100);
		if (last_procent != procent) {
			last_procent = procent;
			PrintProcent(procent);
		}
		//count +=h;
		//cout << setprecision(2) << count << endl;
		for (int j = 1; j < image[0].size() - 1; j++)
		{
			output[i][j] = MedianFilter(image, i, j);
		}
	}
	return output;
}

bool IsCudaSuccess(hipError_t hipError_t, const char* message)
{
	if (hipError_t != hipSuccess) {
		fprintf(stderr, message);
		fprintf(stderr, hipGetErrorString(hipError_t));
		fprintf(stderr, "\n");
		return false;
	}
	return true;
}
int main()
{
	system ("clear");
	draw_frame();
	BMP Input;
	Input.ReadFromFile("input.bmp");
	int width = Input.TellWidth();
	int height = Input.TellHeight();

	vector<vector<int>> a(width + 2, vector <int>(height + 2));

	// convert each pixel to grayscale using RGB->YUV
	for (int j = 0; j < height; j++)
	{
		for (int i = 0; i < width; i++)
		{
			int Temp = (int)floor(0.299 * Input(i, j)->Red + 0.587 * Input(i, j)->Green + 0.114 * Input(i, j)->Blue);
			a[i + 1][j + 1] = Temp;
		}
	}

	for (size_t j = 1; j < height - 1; j++)
	{
		a[0][j] = a[1][j];
		a[width - 1][j] = a[width - 2][j];
	}
	for (size_t i = 1; i < width - 1; i++)
	{
		a[i][0] = a[i][1];
		a[i][height - 1] = a[i][height - 2];
	}
	a[0][0] = a[1][1];
	a[0][height - 1] = a[1][height - 2];
	a[width - 1][0] = a[width - 2][1];
	a[width - 1][height - 1] = a[width - 2][height - 2];

	float* h_data = (float*)malloc(width * height * sizeof(float));
	for (int i = 1; i < width+1; ++i)
		for (int j = 1; j < height+1; ++j)
			h_data[i * height + j] = a[i+1][j+1];

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipArray_t arrayInput;
	float* arrayOutput;

	hipError_t cuerr = hipMalloc((void**)&arrayOutput, width * height * sizeof(float));
	if (!IsCudaSuccess (cuerr, "Cannot allocate device Ouput array for a: ")) return 0;

	cuerr = hipMallocArray(&arrayInput, &channelDesc, width, height);
	if (!IsCudaSuccess (cuerr, "Cannot allocate device Input array for a: ")) return 0;

	cuerr = hipMemcpy2DToArray(arrayInput, 0, 0, h_data, (width) * sizeof(float), (width) * sizeof(float), (height), hipMemcpyHostToDevice);
	if (!IsCudaSuccess (cuerr, "Cannot copy a array2D from host to device: ")) return 0;

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = arrayInput;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeBorder; // ����� Wrap
	texDesc.addressMode[1] = hipAddressModeBorder;
	//texDesc.filterMode = hipFilterModeLinear; // ��������� ��������
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0; // �� ������������ ��������������� ���������

	// Create texture object
	hipTextureObject_t texObj = 0;
	cuerr = hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	if (!IsCudaSuccess (cuerr, "Cannot create TextureObject: ")) return 0;

	// �������� ������������ �������
  	hipEvent_t start, stop;
  	float gpuTime = 0.0f;
  	cuerr = hipEventCreate(&start);
	if (!IsCudaSuccess(cuerr, "Cannot create CUDA start event: ")) return 0;

   	cuerr = hipEventCreate(&stop);
	if (!IsCudaSuccess(cuerr, "Cannot create CUDA end event: ")) return 0;

	dim3 BLOCK_SIZE(32, 32, 1);
	dim3 GRID_SIZE(height  / 32 + 1, width/ 32 + 1, 1);

	// ��������� ����� ������
    	cuerr = hipEventRecord(start, 0);
    	if (cuerr != hipSuccess) {
        	fprintf(stderr, "Cannot record CUDA event: %s\n",
            	hipGetErrorString(cuerr));
        	return 0;
    	}

	kernel << <GRID_SIZE, BLOCK_SIZE >> > (arrayOutput, texObj, width, height);

	cuerr = hipGetLastError();
	if (!IsCudaSuccess (cuerr, "Cannot launch CUDA kernel: ")) return 0;

	// ������������� ���������
	cuerr = hipDeviceSynchronize();
	if (!IsCudaSuccess (cuerr, "Cannot synchronize CUDA kernel: ")) return 0;

	cuerr = hipEventRecord(stop, 0);
	if (!IsCudaSuccess(cuerr, "Cannot copy c array from device to host: ")) return 0;

	cuerr = hipMemcpy(h_data, arrayOutput, width * sizeof(float) * height, hipMemcpyDeviceToHost);
	if (!IsCudaSuccess(cuerr, "Cannot copy a array from device to host: ")) return 0;	

	struct timespec mt1, mt2; 
  	long double tt;
	clock_gettime(CLOCK_REALTIME, &mt1);

	a = transformationImage(a);

	clock_gettime(CLOCK_REALTIME, &mt2);
  	tt=1000000000*(mt2.tv_sec - mt1.tv_sec)+(mt2.tv_nsec - mt1.tv_nsec);
  	cout << "Time CPU: " << tt/1000000000  << " second"<< endl;
	
  	cuerr = hipEventElapsedTime(&gpuTime, start, stop);
  	cout << "Time GPU: " << gpuTime /1000 << " second" << endl;
	cout << "SpeedUp: " << tt/(gpuTime*1000000) << endl;
	cout << "Width: " << width << endl;
	cout << "Height: "<< height << endl;

	for (int j = 0; j < height; j++)
	{
		for (int i = 0; i < width; i++)
		{	//a[i + 1][j + 1]
			ebmpBYTE color = (ebmpBYTE)h_data[i * height + j];
			Input(i, j)->Red = color;
			Input(i, j)->Green = color;
			Input(i, j)->Blue = color;
		}
	}
	BMP Output;
	Output.ReadFromFile("input.bmp");
		
	for (int j = 0; j < height; j++)
	{
		for (int i = 0; i < width; i++)
		{
			ebmpBYTE color = (ebmpBYTE)a[i + 1][j + 1];
			Output(i, j)->Red = color;
			Output(i, j)->Green = color;
			Output(i, j)->Blue = color;
		}
	}
	
	Input.WriteToFile("outputGPU.bmp");
	Output.WriteToFile("outputCPU.bmp");
	
	hipDestroyTextureObject(texObj);
	hipFreeArray(arrayInput);
	hipFree(arrayOutput);
	
	free(h_data);
	return 0;
}